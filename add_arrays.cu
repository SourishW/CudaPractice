#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
__global__ void vectorAddKernel(int* A, int* B, int*C, int N){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N){
        C[index] = A [index] + B[index];
    }
    printf("Thread %d, BlockIdx %d, Dim %d: index: %d\n A: %d, B: %d, C: %d\n", threadIdx.x, blockIdx.x, blockDim.x, index, A[index], B[index], C[index]);
    
}

int main() {
    int N = 1024;
    int size = N * sizeof(int);

    int* A,* B, *C;
    int *d_A, *d_B, *d_C;

    A = new int[N];
    B = new int[N];
    C = new int[N];

    for (int i = 0; i< N; i++){
        A[i] = i;
        B[i] = N-i;
    }

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock -1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    for (int i = 0; i< N; i++){
        if (C[i] != N){
            std::cout << "Problem: "<< C[i] <<" is not " << N << std::endl;
        }
    }

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;

}
